#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/unique.h>
#include <thrust/tuple.h>
#include <hipcub/hipcub.hpp>
#include "../include/voxel_downsample.cuh"
#include "../include/hash_grid.cuh"

#include <cstdint>
#include <iostream>
#include <vector>

void cuda_vgicp::voxelgrid_downsample(
    const float3* __restrict__ d_points,
    size_t N,
    float leaf_size,
    int* __restrict__ d_num_unique_cells
) {
    // Set maximum block and thread properties
    int threadsPerBlock, minBlocksCent;
    hipOccupancyMaxPotentialBlockSize(
        &minBlocksCent,
        &threadsPerBlock,
        init_indices_and_compute_cell_hashes_kernel,
        0,
        0
    );
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;


    // Used device pointer
    int* d_point_indices;
    uint32_t* d_point_cell_hashes;
    uint32_t* d_unique_point_cell_hashes;
    int* d_cell_start;
    int* d_cell_end;
    int* d_points_per_cell;
    int* d_neighbor_indices;
    float* d_point_covariances;
    float* d_voxel_centroids;
    float* d_voxel_covariances;

    hipMalloc(&d_point_indices, N * sizeof(int));
    hipMalloc(&d_point_cell_hashes, N * sizeof(uint32_t));
    hipMemset(d_point_cell_hashes, 0, N * sizeof(uint32_t));
    hipMalloc(&d_unique_point_cell_hashes, N * sizeof(uint32_t));
    hipMalloc(&d_cell_start, N * sizeof(int));
    hipMalloc(&d_cell_end, N * sizeof(int));
    hipMalloc(&d_points_per_cell, N * sizeof(int));
    hipMalloc(&d_neighbor_indices, N * K_NEIGHBORS * sizeof(int));
    hipMalloc(&d_point_covariances, N * 6 * sizeof(float));
    hipMalloc(&d_voxel_centroids, N * 3 * sizeof(float));
    hipMalloc(&d_voxel_covariances, N * 6 * sizeof(float));


    init_indices_and_compute_cell_hashes_kernel<<<blocks, threadsPerBlock>>>(
        d_points,
        N,
        leaf_size,
        d_point_indices,
        d_point_cell_hashes
    );

    /// 0.74 msec/scan

    // Sort point indices by cell hash
    size_t temp_bytes = 0;
    void* d_temp = nullptr;
    hipcub::DeviceRadixSort::SortPairs(
        d_temp,
        temp_bytes,
        d_point_cell_hashes,
        d_point_cell_hashes,
        d_point_indices,
        d_point_indices,
        N
    );
    hipMalloc(&d_temp, temp_bytes);
    hipcub::DeviceRadixSort::SortPairs(
        d_temp,
        temp_bytes,
        d_point_cell_hashes,
        d_point_cell_hashes,
        d_point_indices,
        d_point_indices,
        N
    );
    hipFree(d_temp);

    // Get amount of unique cells
    size_t rle_temp_bytes = 0;
    void* rle_temp = nullptr;
    hipcub::DeviceRunLengthEncode::Encode(
        rle_temp, rle_temp_bytes,
        d_point_cell_hashes,
        d_unique_point_cell_hashes,
        d_points_per_cell,
        d_num_unique_cells,
        N
    );
    hipMalloc(&rle_temp, rle_temp_bytes);
    hipcub::DeviceRunLengthEncode::Encode(
        rle_temp, rle_temp_bytes,
        d_point_cell_hashes,
        d_unique_point_cell_hashes,
        d_points_per_cell,
        d_num_unique_cells,
        N
    );
    hipFree(rle_temp);

    /// 0.93 msec/scan

    compute_cell_ranges_kernel<<<blocks, threadsPerBlock>>>(
        d_point_cell_hashes,
        d_cell_start,
        d_cell_end,
        d_unique_point_cell_hashes,
        N
    );

    /// 0.93 msec/scan

    find_k_nearest_neighbors_kernel<<<blocks, threadsPerBlock>>>(
        d_points,
        N,
        d_point_indices,
        d_unique_point_cell_hashes,
        d_cell_start,
        d_cell_end,
        d_num_unique_cells,
        d_points_per_cell,
        leaf_size,
        d_neighbor_indices
    );

    /// 20 Nachbarn
    /// 111.4 - 1.7     msec/scan --- BOTTLENECK
    /// 104.2 - 1.65    msec/scan --- BOTTLENECK (ohne iterativer Ansatz)
    /// 72.69 - 2.2     msec/scan --- knn - Fallback mit nur einer Zelle

    compute_point_covariances_kernel<<<blocks, threadsPerBlock>>>(
        d_points,
        N,
        d_neighbor_indices,
        d_point_covariances
    );

    compute_voxel_means_and_covariances_kernel<<<blocks, threadsPerBlock>>>(
        d_points,
        d_cell_start,
        d_cell_end,
        d_num_unique_cells,
        d_point_covariances,
        d_voxel_centroids,
        d_voxel_covariances
    );

    hipFree(d_point_indices);
    hipFree(d_point_cell_hashes);
    hipFree(d_unique_point_cell_hashes);
    hipFree(d_cell_start);
    hipFree(d_cell_end);
    hipFree(d_points_per_cell);
    hipFree(d_neighbor_indices);
    hipFree(d_point_covariances);
    hipFree(d_voxel_centroids);
    hipFree(d_voxel_covariances);
}
